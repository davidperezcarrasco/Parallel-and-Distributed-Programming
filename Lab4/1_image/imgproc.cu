/*
 *     
 *  IMAGE PROCESSING
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"

#define pixel(i, j, n)  (((j)*(n)) +(i))


/*read*/
void  readimg(char * filename,int nx, int ny, int * image){
  
   FILE *fp=NULL;

   fp = fopen(filename,"r");
   for(int j=0; j<ny; ++j){
      for(int i=0; i<nx; ++i){
         fscanf(fp,"%d", &image[pixel(i,j,nx)]);      
      }
   }
   fclose(fp);
}

/* save */   
void saveimg(char *filename,int nx,int ny,int *image){

   FILE *fp=NULL;
   fp = fopen(filename,"w");
   for(int j=0; j<ny; ++j){
      for(int i=0; i<nx; ++i){
         fprintf(fp,"%d ", image[pixel(i,j,nx)]);      
      }
      fprintf(fp,"\n");
   }
   fclose(fp);

}

/*invert*/
__global__ void invert(int* image, int* image_invert, int nx, int ny){
   int indx = pixel(threadIdx.x, blockIdx.x, blockDim.x);
   int indy = pixel(threadIdx.y, blockIdx.y, blockDim.x);

   if (indx >= 0 && indx <= nx){
      if (indy >= 0 && indy <= ny){
         image_invert[(pixel(indx, indy, nx))] = 255 - image[(pixel(indx, indy, nx))];
      }
   }
}

/*smooth*/
__global__ void smooth(int* image, int* image_smooth, int nx, int ny){
   int indx = pixel(threadIdx.x, blockIdx.x, blockDim.x);
   int indy = pixel(threadIdx.y, blockIdx.y, blockDim.x);

   if (indx >= 0 && indx <= nx - 1){
      if (indy >= 0 && indy <= ny - 1){
         if (indx !=0 && indy != 0 && indx != nx - 1 && indy != ny - 1){
            image_smooth[pixel(indx, indy, nx)] = image[pixel(indx-1, indy-1, nx)];
            image_smooth[pixel(indx, indy, nx)] += image[pixel(indx-1, indy, nx)];
            image_smooth[pixel(indx, indy, nx)] += image[pixel(indx-1, indy+1, nx)];
            image_smooth[pixel(indx, indy, nx)] += image[pixel(indx, indy-1, nx)];
            image_smooth[pixel(indx, indy, nx)] += image[pixel(indx, indy, nx)];
            image_smooth[pixel(indx, indy, nx)] += image[pixel(indx, indy+1, nx)];
            image_smooth[pixel(indx, indy, nx)] += image[pixel(indx+1, indy-1, nx)];
            image_smooth[pixel(indx, indy, nx)] += image[pixel(indx+1, indy, nx)];
            image_smooth[pixel(indx, indy, nx)] += image[pixel(indx+1, indy+1, nx)];
            image_smooth[pixel(indx, indy, nx)] = 1.0/9.0 * image_smooth[pixel(indx, indy, nx)];
         }
         else image_smooth[pixel(indx, indy, nx)] = 0;
         
      }
   }
}

/*detect*/
__global__ void detect(int* image, int* image_detect, int nx, int ny){
   int indx = pixel(threadIdx.x, blockIdx.x, blockDim.x);
   int indy = pixel(threadIdx.y, blockIdx.y, blockDim.x);

   if (indx >= 0 && indx <= nx - 1){
      if (indy >= 0 && indy <= ny - 1){
         if (indx !=0 && indy != 0 && indx != nx - 1 && indy != ny - 1){
            image_detect[pixel(indx, indy, nx)] = image[pixel(indx-1, indy, nx)];
            image_detect[pixel(indx, indy, nx)] += image[pixel(indx+1, indy, nx)];
            image_detect[pixel(indx, indy, nx)] += image[pixel(indx, indy-1, nx)];
            image_detect[pixel(indx, indy, nx)] += image[pixel(indx, indy+1, nx)];
            image_detect[pixel(indx, indy, nx)] -= 4*image[pixel(indx, indy, nx)];
         
         //correct values that are out of bounds 
         if (image_detect[pixel(indx, indy, nx)] > 255) image_detect[pixel(indx, indy, nx)] = 255;
         else if (image_detect[pixel(indx, indy, nx)] < 0) image_detect[pixel(indx, indy, nx)] = 0;
         }
         else image_detect[pixel(indx, indy, nx)] = 0;
         
      }
   }
}

/*enhance*/
__global__ void enhance(int* image,int *image_enhance,int nx, int ny){
   int indx = pixel(threadIdx.x, blockIdx.x, blockDim.x);
   int indy = pixel(threadIdx.y, blockIdx.y, blockDim.x);

   if (indx >= 0 && indx <= nx - 1){
      if (indy >= 0 && indy <= ny - 1){
         if (indx !=0 && indy != 0 && indx != nx - 1 && indy != ny - 1){
            image_enhance[pixel(indx, indy, nx)] = -image[pixel(indx-1, indy, nx)];
            image_enhance[pixel(indx, indy, nx)] -= image[pixel(indx+1, indy, nx)];
            image_enhance[pixel(indx, indy, nx)] -= image[pixel(indx, indy-1, nx)];
            image_enhance[pixel(indx, indy, nx)] -= image[pixel(indx, indy+1, nx)];
            image_enhance[pixel(indx, indy, nx)] += 5*image[pixel(indx, indy, nx)]; 
            
            //correct values that are out of bounds 
            if (image_enhance[pixel(indx, indy, nx)] > 255) image_enhance[pixel(indx, indy, nx)] = 255;
            else if (image_enhance[pixel(indx, indy, nx)] < 0) image_enhance[pixel(indx, indy, nx)] = 0;
         }
         else image_enhance[pixel(indx, indy, nx)] = 0;
         
      }
   }

}

/* Main program */
int main (int argc, char *argv[])
{
   int    nx,ny;
   char   filename[250];
   int B_size = 16;

   /* Get parameters */
   if (argc != 4) 
   {
      printf ("Usage: %s image_name N M \n", argv[0]);
      exit (1);
   }
   sprintf(filename, "%s.txt", argv[1]);
   nx  = atoi(argv[2]);
   ny  = atoi(argv[3]);

   printf("%s %d %d\n", filename, nx, ny);

   /* Allocate CPU and GPU pointers */

   int*   image=(int *) malloc(sizeof(int)*nx*ny); 
   int*   image_invert  = (int *) malloc(sizeof(int)*nx*ny);  
   int*   image_smooth  = (int *) malloc(sizeof(int)*nx*ny);  
   int*   image_detect  = (int *) malloc(sizeof(int)*nx*ny);  
   int*   image_enhance = (int *) malloc(sizeof(int)*nx*ny);

   int*   d_image;
   int*   d_image_invert;
   int*   d_image_smooth;
   int*   d_image_detect;
   int*   d_image_enhance; 

   hipMalloc((void **)&d_image, nx*ny*sizeof(int));
   hipMalloc((void **)&d_image_invert, nx*ny*sizeof(int));
   hipMalloc((void **)&d_image_smooth, nx*ny*sizeof(int));
   hipMalloc((void **)&d_image_detect, nx*ny*sizeof(int));
   hipMalloc((void **)&d_image_enhance, nx*ny*sizeof(int));
  

   /* Read image and save in array imgage */
   readimg(filename,nx,ny,image);


  /* Print runtime */
   dim3 dimBlock(B_size, B_size, 1);
   int dimgx = (nx + B_size - 1) / B_size;
   int dimgy = (ny + B_size - 1) / B_size;
   dim3 dimGrid(dimgx, dimgy, 1);

   float runtime;
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   hipEventRecord(start);
   hipMemcpy(d_image, image, sizeof(int) * nx * ny, hipMemcpyHostToDevice);
   
   invert<<<dimGrid, dimBlock, sizeof(int) * B_size * B_size>>>(d_image, d_image_invert, nx, ny);
   hipMemcpy(image_invert, d_image_invert, sizeof(int) * nx * ny, hipMemcpyDeviceToHost);

   smooth<<<dimGrid, dimBlock, sizeof(int) * B_size * B_size>>>(d_image, d_image_smooth, nx, ny);
   hipMemcpy(image_smooth, d_image_smooth, sizeof(int) * nx * ny, hipMemcpyDeviceToHost);

   enhance<<<dimGrid, dimBlock, sizeof(int) * B_size * B_size>>>(d_image, d_image_detect, nx, ny);
   hipMemcpy(image_enhance, d_image_enhance, sizeof(int) * nx * ny, hipMemcpyDeviceToHost);

   detect<<<dimGrid, dimBlock, sizeof(int) * B_size * B_size>>>(d_image, d_image_enhance, nx, ny);
   hipMemcpy(image_detect, d_image_detect, sizeof(int) * nx * ny, hipMemcpyDeviceToHost);

   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&runtime, start, stop);

   printf("The total time that the data communication and the computation of images is: %lf\n", runtime);

   
   /* Save images */
   char fileout[255]={0};
   sprintf(fileout, "%s-inverse.txt", argv[1]);
   saveimg(fileout,nx,ny,image_invert);
   sprintf(fileout, "%s-smooth.txt", argv[1]);
   saveimg(fileout,nx,ny,image_smooth);
   sprintf(fileout, "%s-detect.txt", argv[1]);
   saveimg(fileout,nx,ny,image_detect);
   sprintf(fileout, "%s-enhance.txt", argv[1]);
   saveimg(fileout,nx,ny,image_enhance);

   /* Deallocate CPU and GPU pointers*/
   free(image);
   free(image_invert);
   free(image_smooth);
   free(image_detect);
   free(image_enhance);

   hipFree(d_image_invert);
   hipFree(d_image_smooth);
   hipFree(d_image_detect);
   hipFree(d_image_enhance);
}